#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cstdlib>
#include <cmath>
#include "PCR_Class.h"
#include "tdma.h"

double maxarr(double *arr, int Nx) {
    double v = arr[0];
    for(int nx=0; nx<Nx; nx++) {
        if (v < arr[nx]) {
            v = arr[nx];
        }
    }
    return v;
}

double minarr(double *arr, int Nx) {
    double v = arr[0];
    for(int nx=0; nx<Nx; nx++) {
        if (v > arr[nx]) {
            v = arr[nx];
        }
    }
    return v;
}

double averarr(double *arr, int Nx) {
    double v = 0.0;
    for(int nx=0; nx<Nx; nx++) {
        v += arr[nx];
    }
    return v / double(Nx);
}

int main( ) {

    std::chrono::time_point<std::chrono::system_clock> tstart, tend;
    std::chrono::duration<double> duration;    
    size_t diagonal_size = 1000;
    
    PCR_Solver crs = PCR_Solver(diagonal_size);
    
    //Generate sampel data
    srand (time(NULL));
    
    thrust::device_vector<float> alist(diagonal_size);
    thrust::device_vector<float> blist(diagonal_size);
    thrust::device_vector<float> clist(diagonal_size);
    thrust::device_vector<float> dlist(diagonal_size);
    thrust::device_vector<float> xlist(diagonal_size);
    
    float * ptr_alist = thrust::raw_pointer_cast(alist.data());
    float * ptr_blist = thrust::raw_pointer_cast(blist.data());
    float * ptr_clist = thrust::raw_pointer_cast(clist.data());
    float * ptr_dlist = thrust::raw_pointer_cast(dlist.data());
    float * ptr_xlist = thrust::raw_pointer_cast(xlist.data());

    size_t Nx = diagonal_size;
    float *a, *b, *c, *rhs, *old;
    a = new float[Nx];
    b = new float[Nx];
    c = new float[Nx];
    rhs = new float[Nx];
    old = new float[Nx];

    int trynumber = 100;
    double *gputimes = new double[trynumber];
    double *cputimes = new double[trynumber]; 

    for(int count=0; count<trynumber; count++) {
        for (int i=0; i < diagonal_size; i++) {
            alist[i] = -1.0+0.1*float(rand()) / float(RAND_MAX);
            blist[i] = 2.0+0.1*float(rand()) / float(RAND_MAX);
            clist[i] = -1.0+0.1*float(rand()) / float(RAND_MAX);
            dlist[i] = 1.0 + 10.0*float(rand()) / float(RAND_MAX);
            xlist[i] = 0.0f;
            a[i] = alist[i];
            b[i] = blist[i];
            c[i] = clist[i];
            rhs[i] = dlist[i];
            old[i] = xlist[i];
        }
    
        a[0] = float(0.0);
        c[Nx-1] = float(0.0);
        alist[0] = float(0.0);
        clist[diagonal_size-1] = float(0.0);
        
        tstart = std::chrono::system_clock::now();
        crs.Solve(ptr_alist, ptr_blist, ptr_clist, ptr_dlist, ptr_xlist);
        tend = std::chrono::system_clock::now();
        duration = tend - tstart;
        gputimes[count] = duration.count();

        tstart = std::chrono::system_clock::now();
        TDMA(a, b, c, rhs, old, Nx);
        tend = std::chrono::system_clock::now();
        duration = tend - tstart;
        cputimes[count] = duration.count();

        for (size_t it=0; it<Nx; it++) {
            if(fabs(old[it] - xlist[it]) > 1.0e-6) {
                std::cout << old[it] << " " << xlist[it] << ": " << fabs(old[it] - xlist[it]) << std::endl;
            }
        }
    }

    std::cout << "maximum: " << maxarr(gputimes, trynumber) << " <> " << maxarr(cputimes, trynumber) << std::endl;
    std::cout << "minimum: " << minarr(gputimes, trynumber) << " <> " << minarr(cputimes, trynumber) << std::endl;
    std::cout << "average: " << averarr(gputimes, trynumber) << " <> " << averarr(cputimes, trynumber) << std::endl;

    delete []a;
    delete []b;
    delete []c;
    delete []rhs;
    delete []old;
    delete []gputimes;
    delete []cputimes;

    return 0;

}
