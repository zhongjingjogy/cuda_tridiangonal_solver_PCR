#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include "PCR_Class.h"
#include "tdma.h"
#include "utils.h"

float GpuResultCheck(float *a, float *b, float *c, float *rhs, float *old, int Nx) {
    float v = -1.0;
    float v1 = fabs(b[0] * old[0] + c[0] * old[1] - rhs[0]);
    if (v < v1) {
        v = v1;
    }
    for (int nx = 1; nx < Nx - 1; nx++) {
        v1 = fabs(a[nx] * old[nx - 1] + b[nx] * old[nx] + c[nx] * old[nx + 1] -
                  rhs[nx]);
        if (v < v1) {
            v = v1;
        }
    }
    v1 = fabs(b[Nx - 1] * old[Nx - 1] + a[Nx - 1] * old[Nx - 2] - rhs[Nx - 1]);
    if (v < v1) {
        v = v1;
    }

    return v;
}

int Test(size_t diagonal_size) {
    std::chrono::time_point<std::chrono::system_clock> tstart, tend;
    std::chrono::duration<double> duration;

    PCR_Solver crs = PCR_Solver(diagonal_size);

    thrust::device_vector<float> alist(diagonal_size);
    thrust::device_vector<float> blist(diagonal_size);
    thrust::device_vector<float> clist(diagonal_size);
    thrust::device_vector<float> dlist(diagonal_size);
    thrust::device_vector<float> xlist(diagonal_size);

    float *ptr_alist = thrust::raw_pointer_cast(alist.data());
    float *ptr_blist = thrust::raw_pointer_cast(blist.data());
    float *ptr_clist = thrust::raw_pointer_cast(clist.data());
    float *ptr_dlist = thrust::raw_pointer_cast(dlist.data());
    float *ptr_xlist = thrust::raw_pointer_cast(xlist.data());

    size_t Nx = diagonal_size;
    double *a, *b, *c, *rhs, *old;
    a = new double[Nx];
    b = new double[Nx];
    c = new double[Nx];
    rhs = new double[Nx];
    old = new double[Nx];

    float *ma, *mb, *mc, *mrhs, *mx;
    ma = new float[Nx];
    mb = new float[Nx];
    mc = new float[Nx];
    mrhs = new float[Nx];
    mx = new float[Nx];

    int trynumber = 100;
    double *gputimes = new double[trynumber];
    double *cputimes = new double[trynumber];
    bool *cpuresultcheck = new bool[trynumber];
    bool *gpuresultcheck = new bool[trynumber];

    for (int count = 0; count < trynumber; count++) {
        for (int i = 0; i < diagonal_size; i++) {
            alist[i] = -1.0 + 0.1 * float(rand()) / float(RAND_MAX);
            blist[i] = 2.0 + 0.1 * float(rand()) / float(RAND_MAX);
            clist[i] = -1.0 + 0.1 * float(rand()) / float(RAND_MAX);
            dlist[i] = 1.0 + 10.0 * float(rand()) / float(RAND_MAX);
            xlist[i] = 0.0f;

            ma[i] = alist[i];
            mb[i] = blist[i];
            mc[i] = clist[i];
            mrhs[i] = dlist[i];
            mx[i] = xlist[i];

            a[i] = double(ma[i]);
            b[i] = double(mb[i]);
            c[i] = double(mc[i]);
            rhs[i] = double(mrhs[i]);
            old[i] = double(mx[i]);
        }

        a[0] = double(0.0);
        c[Nx - 1] = double(0.0);
        alist[0] = float(0.0);
        clist[diagonal_size - 1] = float(0.0);
        ma[0] = float(0.0);
        mc[diagonal_size-1] = float(0.0);

        tstart = std::chrono::system_clock::now();
        crs.Solve(ptr_alist, ptr_blist, ptr_clist, ptr_dlist, ptr_xlist);
        tend = std::chrono::system_clock::now();
        duration = tend - tstart;
        gputimes[count] = duration.count();
        for (int i = 0; i < diagonal_size; i++) {
            mx[i] = xlist[i];
        }
        gpuresultcheck[count] =
            GpuResultCheck(ma, mb, mc, mrhs, mx, diagonal_size) < 1.0e-4;

        tstart = std::chrono::system_clock::now();
        TDMA<double>(a, b, c, rhs, old, Nx);
        tend = std::chrono::system_clock::now();
        duration = tend - tstart;
        cputimes[count] = duration.count();
        cpuresultcheck[count] = CpuResultCheck(a, b, c, rhs, old, Nx) < 1.0e-10;
    }

    std::cout << "Diagonal size: " << diagonal_size << std::endl;
    std::cout << "Gpu v.s. Cpu" << std::endl;
    std::cout << "Correct results: " << correctcount(gpuresultcheck, trynumber) << " <> " << correctcount(cpuresultcheck, trynumber) << std::endl;
    std::cout << "Maximum time(s): " << maxarr(gputimes, trynumber) << " <> "
              << maxarr(cputimes, trynumber) << std::endl;
    std::cout << "Minimum time(s): " << minarr(gputimes, trynumber) << " <> "
              << minarr(cputimes, trynumber) << std::endl;
    std::cout << "Average time(s): " << averarr(gputimes, trynumber) << " <> "
              << averarr(cputimes, trynumber) << std::endl;
    std::cout << std::endl;

    delete[] a;
    delete[] b;
    delete[] c;
    delete[] rhs;
    delete[] old;
    delete[] ma;
    delete[] mb;
    delete[] mc;
    delete[] mrhs;
    delete[] mx;
    delete[] gputimes;
    delete[] cputimes;
    delete[] cpuresultcheck;
    delete[] gpuresultcheck;

    return 0;
}

int main() {
    // Generate sampel data
    srand(time(NULL));
    std::cout << "Test of the accuracy and efficiency of two algorithms that "
                 "solves the tridiagonal matrix."
              << std::endl;
    Test(10);
    Test(100);
    Test(1000);
    Test(5000);
    return 0;
}
